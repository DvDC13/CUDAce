#include "hip/hip_runtime.h"
/*
    Check if an array is sorted in ascending order
*/

/*
    ./main <size>
*/

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"

__global__ void is_sorted(int* d_in, int* d_out, int size)
{
    extern __shared__ int s_in[];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    int val = 0;
    if (gid < size - 1)
    {
        val = d_in[gid] > d_in[gid + 1] ? 1 : 0;
    }

    s_in[tid] = val;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            s_in[tid] += s_in[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        atomicAdd(d_out, s_in[0]);
    }
}

int main(int argc, char *argv[])
{
    int size = argc > 1 ? atoi(argv[1]) : 1000000;

    int* h_in = new int[size];
    
    for (int i = 0; i < size; i++)
    {
        h_in[i] = i;
    }

    int* d_in;
    hipMalloc(&d_in, size * sizeof(int));
    hipMemcpy(d_in, h_in, size * sizeof(int), hipMemcpyHostToDevice);

    int* d_out;
    hipMalloc(&d_out, sizeof(int));
    hipMemset(d_out, 0, sizeof(int));

    dim3 block(256);
    dim3 grid((size + block.x - 1) / block.x);

    auto start = std::chrono::high_resolution_clock::now();
    is_sorted<<<grid, block, sizeof(int) * block.x>>>(d_in, d_out, size);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;
    printf("GPU time: %f ms\n", duration.count());

    int h_out;
    hipMemcpy(&h_out, d_out, sizeof(int), hipMemcpyDeviceToHost);
    printf("Array is sorted: %s\n", h_out == 0 ? "true" : "false");

    hipFree(d_in);

    delete[] h_in;

    hipDeviceReset();

    return EXIT_SUCCESS;
}