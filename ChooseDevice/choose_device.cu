/*
    Choose the device based on the needs
*/

/*
    ./main
*/

#include <stdio.h>
#include <hip/hip_runtime.h>


int main()
{
    hipDeviceProp_t prop;
    int dev;

    hipGetDevice(&dev);
    
    printf("ID of current CUDA device: %d\n", dev);

    memset(&prop, 0, sizeof(hipDeviceProp_t));
    prop.major = 7;
    prop.minor = 5;

    hipChooseDevice(&dev, &prop);
    printf("ID of CUDA device closest to revision 7.5: %d\n", dev);

    hipSetDevice(dev);

    return EXIT_SUCCESS;
}