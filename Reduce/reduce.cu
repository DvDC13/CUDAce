#include "hip/hip_runtime.h"
/*
    Compute the sum of an array
*/

/*
    ./main <size>
*/

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"

// Interleaved Addressing
__global__ void reduce0(int* d_in, int* d_out, int size)
{
    extern __shared__ int s_in[];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    s_in[tid] = d_in[gid];
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2)
    {
        if (tid % (2 * s) == 0)
        {
            s_in[tid] += s_in[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        atomicAdd(d_out, s_in[0]);
    }
}

// Interleaved Addressing without branch divergence and % operation
__global__ void reduce1(int* d_in, int* d_out, int size)
{
    extern __shared__ int s_in[];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    s_in[tid] = d_in[gid];
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2)
    {
        int index = 2 * s * tid;
        if (index < blockDim.x)
        {
            s_in[index] += s_in[index + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        atomicAdd(d_out, s_in[0]);
    }
}

// Sequence Addressing
__global__ void reduce2(int* d_in, int* d_out, int size)
{
    extern __shared__ int s_in[];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    s_in[tid] = d_in[gid];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            s_in[tid] += s_in[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        atomicAdd(d_out, s_in[0]);
    }
}

// First Add During Load
__global__ void reduce3(int* d_in, int* d_out, int size)
{
    extern __shared__ int s_in[];

    int tid = threadIdx.x;
    int gid = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    s_in[tid] = d_in[gid] + d_in[gid + blockDim.x];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            s_in[tid] += s_in[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        atomicAdd(d_out, s_in[0]);
    }
}

void sum_cpu(int* h_in, int& h_out, int size)
{
    for (int i = 0; i < size; i++)
    {
        h_out += h_in[i];
    }
}

int main(int argc, char** argv)
{
    int size = argc > 1 ? atoi(argv[1]) : 1000000;

    int* h_in = new int[size];
    
    for (int i = 0; i < size; i++)
    {
        h_in[i] = i;
    }

    int h_out_cpu = 0;

    auto start = std::chrono::high_resolution_clock::now();
    sum_cpu(h_in, h_out_cpu, size);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;
    printf("CPU time: %f ms\n", duration.count());

    int* d_in;
    hipMalloc(&d_in, size * sizeof(int));
    hipMemcpy(d_in, h_in, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((size + block.x - 1) / block.x);

    int* d_out_reduce0;
    hipMalloc(&d_out_reduce0, sizeof(int));
    
    // Interleaved Addressing
    start = std::chrono::high_resolution_clock::now();
    reduce0<<<grid, block, sizeof(int) * block.x>>>(d_in, d_out_reduce0, size);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    printf("GPU reduce0 time: %f ms\n", duration.count());

    int* d_out_reduce1;
    hipMalloc(&d_out_reduce1, sizeof(int));

    // Interleaved Addressing without branch divergence and % operation
    start = std::chrono::high_resolution_clock::now();
    reduce1<<<grid, block, sizeof(int) * block.x>>>(d_in, d_out_reduce1, size);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    printf("GPU reduce1 time: %f ms\n", duration.count());

    int* d_out_reduce2;
    hipMalloc(&d_out_reduce2, sizeof(int));

    // Sequence Addressing
    start = std::chrono::high_resolution_clock::now();
    reduce2<<<grid, block, sizeof(int) * block.x>>>(d_in, d_out_reduce2, size);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    printf("GPU reduce2 time: %f ms\n", duration.count());

    int* d_out_reduce3;
    hipMalloc(&d_out_reduce3, sizeof(int));

    grid = (size + block.x * 2 - 1) / (block.x * 2);

    // First Add During Load
    start = std::chrono::high_resolution_clock::now();
    reduce3<<<grid, block, sizeof(int) * block.x * 2>>>(d_in, d_out_reduce3, size);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    printf("GPU reduce3 time: %f ms\n", duration.count());

    int* d_out_reduce0_gpu = new int;
    hipMemcpy(d_out_reduce0_gpu, d_out_reduce0, sizeof(int), hipMemcpyDeviceToHost);

    int* d_out_reduce1_gpu = new int;
    hipMemcpy(d_out_reduce1_gpu, d_out_reduce1, sizeof(int), hipMemcpyDeviceToHost);

    int* d_out_reduce2_gpu = new int;
    hipMemcpy(d_out_reduce2_gpu, d_out_reduce2, sizeof(int), hipMemcpyDeviceToHost);

    int* d_out_reduce3_gpu = new int;
    hipMemcpy(d_out_reduce3_gpu, d_out_reduce3, sizeof(int), hipMemcpyDeviceToHost);

    printf("CPU: %d\n", h_out_cpu);
    printf("GPU reduce0: %d\n", *d_out_reduce0_gpu);
    printf("GPU reduce1: %d\n", *d_out_reduce1_gpu);
    printf("GPU reduce2: %d\n", *d_out_reduce2_gpu);
    printf("GPU reduce3: %d\n", *d_out_reduce3_gpu);

    delete[] h_in;

    hipFree(d_in);
    hipFree(d_out_reduce0);
    hipFree(d_out_reduce1);
    hipFree(d_out_reduce2);
    hipFree(d_out_reduce3);

    return EXIT_SUCCESS; 
}