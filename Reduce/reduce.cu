#include "hip/hip_runtime.h"
/*
    Compute the sum of an array
*/

/*
    ./main <size>
*/

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""
#include "thrust/device_vector.h"

// Interleaved Addressing
__global__ void reduce0(int* d_in, int* d_out, int size)
{
    extern __shared__ int s_in[];

    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    s_in[tid] = d_in[gid];
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2)
    {
        if (tid % (2 * s) == 0)
        {
            s_in[tid] += s_in[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        atomicAdd(d_out, s_in[0]);
    }
}

void sum_cpu(int* h_in, int& h_out, int size)
{
    for (int i = 0; i < size; i++)
    {
        h_out += h_in[i];
    }
}

int main(int argc, char** argv)
{
    int size = argc > 1 ? atoi(argv[1]) : 1000000;

    int* h_in = new int[size];
    
    for (int i = 0; i < size; i++)
    {
        h_in[i] = i;
    }

    int h_out_cpu = 0;

    auto start = std::chrono::high_resolution_clock::now();
    sum_cpu(h_in, h_out_cpu, size);
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<float, std::milli> duration = end - start;
    printf("CPU time: %f ms\n", duration.count());

    int* d_in;
    int* d_out;

    hipMalloc(&d_in, size * sizeof(int));
    hipMalloc(&d_out, sizeof(int));

    hipMemcpy(d_in, h_in, size * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(256);
    dim3 grid((size + block.x - 1) / block.x);
    
    start = std::chrono::high_resolution_clock::now();
    reduce0<<<grid, block, sizeof(int) * block.x>>>(d_in, d_out, size);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    printf("GPU time: %f ms\n", duration.count());

    int* d_out_gpu = new int;
    hipMemcpy(d_out_gpu, d_out, sizeof(int), hipMemcpyDeviceToHost);

    printf("CPU: %d\n", h_out_cpu);
    printf("GPU: %d\n", *d_out_gpu);

    delete[] h_in;

    return EXIT_SUCCESS; 
}